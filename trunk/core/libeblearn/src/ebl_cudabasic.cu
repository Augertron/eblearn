#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2011 by Soumith Chintala*
 *   soumith@gmail.com  *
 *   All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Redistribution under a license not approved by the Open Source
 *       Initiative (http://www.opensource.org) must display the
 *       following acknowledgement in all advertising material:
 *        This product includes software developed at the Courant
 *        Institute of Mathematical Sciences (http://cims.nyu.edu).
 *     * The names of the authors may not be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED ``AS IS'' AND ANY EXPRESS OR IMPLIED
 * WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL ThE AUTHORS BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ***************************************************************************/

#include "ebl_cudabasic.h"

#ifdef __CUDA__

namespace ebl {

#define CUDA_SHARED_MEM_SIZE 0
  //(4*1024-32) 
// this is given by nVidia: max shared mem per block

/*
 * Description:
 *   base conv2D routine: 3D input, 3D output, 4D kernel
 *
 *   - all chunks of data should be contiguous
 *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
 *   - the templated kernel size is useful to generate code that's 2x faster
 *     but can be set to 0 to allow arbitrary kernel sizes
 */
template <bool swapkernel, int T_kernel_h, int T_kernel_w>
  __global__ void conv2generic(float *input, float *kernel, float *output,
                               int input_n, int input_h, int input_w,
                               int kernel_n, int kernel_h, int kernel_w,
                               int stride_h, int stride_w)
{
  // output dimensions
  int output_h = (input_h - kernel_h) / stride_h + 1;
  int output_w = (input_w - kernel_w) / stride_w + 1;

  // xcorr or conv
  int koffset = swapkernel ? kernel_w*kernel_h-1 : 0;

  // nb outputs
  int output_n = kernel_n / input_n;

  // generate offsets according to block/thread ids
  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  int oo_start = blockIdx.x;
  int oo_end = oo_start+1;

  int ii_start = (blockIdx.x / output_n) * input_n;
  int ii_end = ii_start + input_n;

  // nb threads, unique thread id
  int tid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;
  int nthreads = blockDim.x * blockDim.y * blockDim.z;

  // iterators
  int oo, ii, xx, yy, kx, ky, kk;

  // do the kernels fit in shared mem ?
  if (input_n*kernel_w*kernel_h <= CUDA_SHARED_MEM_SIZE) {

    // put the kernel in shared memory
    __shared__ float shared_kernel[CUDA_SHARED_MEM_SIZE];

    // first thread of each block does the copy
    for (kk = tid; kk < kernel_w*kernel_h*input_n; kk += nthreads) {
      shared_kernel[kk] = kernel[input_n*kernel_w*kernel_h*(oo_start % output_n) + kk];
    }
    __syncthreads();

    // templated kernel size
    if ((T_kernel_w > 0) && (T_kernel_h > 0)) {
      // unrolled convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for(ii = ii_start; ii < ii_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = shared_kernel + (ii % input_n)*kernel_w*kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    } else {
      // default convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for(ii = ii_start; ii < ii_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = shared_kernel + (ii % input_n) * kernel_w * kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    }

  } else { // not enough shared mem for kernels, simply stream them

    // convolution loop
    for(oo = oo_start; oo < oo_end; oo++) {
      for(ii = ii_start; ii < ii_end; ii++) {
        for(yy = yy_start; yy < yy_end; yy+=yy_step) {
          for(xx = xx_start; xx < xx_end; xx+=xx_step) {
            // Dot product in two dimensions... (between input image and the mask)
            float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
            float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
            float *kernel_p = kernel + ((oo % output_n) * input_n + (ii % input_n))*kernel_w*kernel_h + koffset;
            float sum = 0;
            if (swapkernel) {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p--);
                }
                input_p += input_w;
              }
            } else {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p++);
                }
                input_p += input_w;
              }
            }
            *output_p += sum;
          }
        }
      }
    }
  }
}

  void ebl::cuda_convolution_3d(idx<float32> &in, idx<float32> &ker,
			      idx<float32> &out, 
                         intg stride_x, intg stride_y, int devid) {
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input and kernels to gpu, allocate output on gpu
    hipError_t err;
    float *input_data, *kernel_data, *output_data;
    LOCAL_TIMING_START();
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &kernel_data, ker.nelements() * sizeof(float));
    hipMemcpy( kernel_data, ker.idx_ptr(), ker.nelements() * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &output_data, out.nelements() * sizeof(float));
    hipMemcpy(output_data, out.idx_ptr(), out.nelements() * sizeof(float), hipMemcpyHostToDevice);
    // set the number of blocks and threads
    int nthreads_x = 32;
    int nthreads_y = 8;
    int block_height = floor(16 / out.dim(0));
    if (block_height < 1)
      block_height = 1;
    dim3 blocks(out.dim(0),block_height);
    dim3 threads(nthreads_x,nthreads_y);

    // sync any previous kernel exec
    hipDeviceSynchronize();
    LOCAL_TIMING_REPORT("convgpu initial transfer time" << in.dim(0));
    LOCAL_TIMING2_START();
    if ((ker.dim(2) == 3) && (ker.dim(1) == 3))
      conv2generic <false, 3, 3> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 5) && (ker.dim(1) == 5))
      conv2generic <false, 5, 5> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 7) && (ker.dim(1) == 7))
      conv2generic <false, 7, 7> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 9) && (ker.dim(1) == 9))
      conv2generic <false, 9, 9> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 11) && (ker.dim(1) == 11))
      conv2generic <false, 11, 11> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else if ((ker.dim(2) == 13) && (ker.dim(1) == 13))
      conv2generic <false, 13, 13> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else if ((ker.dim(2) == 4) && (ker.dim(1) == 4))
      conv2generic <false, 4, 4> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 6) && (ker.dim(1) == 6))
      conv2generic <false, 6, 6> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 8) && (ker.dim(1) == 8))
      conv2generic <false, 8, 8> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y);
    else if ((ker.dim(2) == 10) && (ker.dim(1) == 10))
      conv2generic <false, 10, 10> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else if ((ker.dim(2) == 12) && (ker.dim(1) == 12))
      conv2generic <false, 12, 12> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y);
    else
      conv2generic <false, 0 , 0> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                         in.dim(0), in.dim(1), in.dim(2),
                                                         out.dim(0)*in.dim(0), ker.dim(1), ker.dim(2),
                                                         stride_x, stride_y);



  LOCAL_TIMING2_REPORT("convgpu kernel execution time");
  // sync & clean
  hipDeviceSynchronize();
  hipMemcpy(out.idx_ptr(), output_data, out.nelements() * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(input_data);
  hipFree(kernel_data);
  hipFree(output_data);

  // check for errors
  err = hipGetLastError();
  if (err != hipSuccess)
    eblerror("cuda error" << hipGetErrorString(err));
}


  /////////////////////////////////////////////////////////////////////////////
  //////////////////////
  /// ebl_convolution_3dmap
  /////////////////////////////////////////////////////////////////////////////
  /*
   * Description:
   *   base conv2D routine: 3D input, 3D output, 4D kernel
   *
   *   - all chunks of data should be contiguous
   *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
   *   - the templated kernel size is useful to generate code that's 2x faster
   *     but can be set to 0 to allow arbitrary kernel sizes
   *   ---- the table should have the first dim with the outputs, each output 
   *   ---- should have a fanin set of inputs contiguously
   */
  template <bool swapkernel, int T_kernel_h, int T_kernel_w>
  __global__ void conv2mapgeneric(float *input, float *kernel, float *output,
                               int input_n, int input_h, int input_w,
                               int kernel_n, int kernel_h, int kernel_w,
                                  int stride_h, int stride_w,
                                  long *table, int fanin)
  {
    // output dimensions
    int output_h = (input_h - kernel_h) / stride_h + 1;
    int output_w = (input_w - kernel_w) / stride_w + 1;

    // xcorr or conv
    int koffset = swapkernel ? kernel_w*kernel_h-1 : 0;

    // nb outputs
    int output_n = kernel_n / fanin;

    // generate offsets according to block/thread ids
    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y*gridDim.y;

    int oo_start = blockIdx.x;
    int oo_end = oo_start+1;

    int table_start = blockIdx.x * (fanin * 2);
    int table_end = table_start + (fanin * 2);

    // nb threads, unique thread id
    int tid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;
    int nthreads = blockDim.x * blockDim.y * blockDim.z;

    // iterators
    int oo, ii, xx, yy, kx, ky, kk;

    // do the kernels fit in shared mem ?
    if (fanin*kernel_w*kernel_h <= CUDA_SHARED_MEM_SIZE) {

      // put the kernel in shared memory
      __shared__ float shared_kernel[CUDA_SHARED_MEM_SIZE];

      // first thread of each block does the copy
      for (kk = tid; kk < kernel_w*kernel_h*fanin; kk += nthreads) {
        shared_kernel[kk] = kernel[fanin*kernel_w*kernel_h*(oo_start % output_n) + kk];
      }
      __syncthreads();

      // templated kernel size
      if ((T_kernel_w > 0) && (T_kernel_h > 0)) {
        // unrolled convolution loop
        for(oo = oo_start; oo < oo_end; oo++) {
          for (ii = table_start; ii < table_end; ii = ii + 2) {
            for(yy = yy_start; yy < yy_end; yy+=yy_step) {
              for(xx = xx_start; xx < xx_end; xx+=xx_step) {
                // Dot product in two dimensions... (between input image and the mask)
                float *input_p = input + table[ii]*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
                float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
                // float *kernel_p = shared_kernel + (table[ii] % fanin)*kernel_w*kernel_h + koffset;
                float *kernel_p = shared_kernel + table[ii + 1] *kernel_w*kernel_h + koffset;
                float sum = 0;
                if (swapkernel) {
#pragma unroll
                  for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                    for(kx = 0; kx < T_kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p--);
                    }
                    input_p += input_w;
                  }
                } else {
#pragma unroll
                  for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                    for(kx = 0; kx < T_kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p++);
                    }
                    input_p += input_w;
                  }
                }
                *output_p += sum;
              }
            }
          }
        }
      } else {
        // default convolution loop
        for(oo = oo_start; oo < oo_end; oo++) {
          for (ii = table_start; ii < table_end; ii++) {
            for(yy = yy_start; yy < yy_end; yy+=yy_step) {
              for(xx = xx_start; xx < xx_end; xx+=xx_step) {
                // Dot product in two dimensions... (between input image and the mask)
                float *input_p = input + table[ii]*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
                float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
                float *kernel_p = shared_kernel + (table[ii] % fanin) * kernel_w * kernel_h + koffset;
                float sum = 0;
                if (swapkernel) {
                  for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                    for(kx = 0; kx < kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p--);
                    }
                    input_p += input_w;
                  }
                } else {
                  for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                    for(kx = 0; kx < kernel_w; kx++) {
                      sum += input_p[kx]*(*kernel_p++);
                    }
                    input_p += input_w;
                  }
                }
                *output_p += sum;
              }
            }
          }
        }
      }

    } else { // not enough shared mem for kernels, simply stream them

      // convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for (ii = table_start; ii < table_end; ii = ii + 2) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + table[ii]*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              //float *kernel_p = kernel + ((oo % output_n) * fanin + (table[ii + 1] % fanin))*kernel_w*kernel_h + koffset;
              float *kernel_p = kernel + table[ii + 1] *kernel_w*kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    }
  }


  void ebl::cuda_convolution_3dmap(idx<float32> &in, idx<float32> &ker,
			      idx<float32> &out, 
                                   intg stride_x, intg stride_y, 
                                   idx<intg> table, int fanin, int devid) {
    // cout << "Executing on Device " << devid << " (CUDA)"<<endl;
    if (devid != -1)
      hipSetDevice(devid);
    // copy input and kernels to gpu, allocate output on gpu
    hipError_t err;
    float *input_data, *kernel_data, *output_data;
    long *table_data;
    LOCAL_TIMING_START();
    hipMalloc((void**) &input_data, in.nelements() * sizeof(float));
    hipMemcpy(input_data, in.idx_ptr(), in.nelements() * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &kernel_data, ker.nelements() * sizeof(float));
    hipMemcpy( kernel_data, ker.idx_ptr(), ker.nelements() * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &output_data, out.nelements() * sizeof(float));
    hipMemcpy(output_data, out.idx_ptr(), out.nelements() * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &table_data, table.nelements() * sizeof(intg));
    hipMemcpy(table_data, table.idx_ptr(), table.nelements() * sizeof(intg), hipMemcpyHostToDevice);
    // set the number of blocks and threads
    int nthreads_x = 32;
    int nthreads_y = 8;
    int block_height = floor(16 / out.dim(0));
    if (block_height < 1)
      block_height = 1;
    dim3 blocks(out.dim(0),block_height);
    dim3 threads(nthreads_x,nthreads_y);
    
    // cout << "DEBUG INFO:" <<"\nnthread_x:" << nthreads_x
    //      <<"\nnthread_y:" << nthreads_y
    //      <<"\nblock_height:" << block_height
    //      <<"\nblock_width:" << out.dim(0)
    //      <<"\tinput_elements: "<<in.nelements()
    //      <<"\tkernel_elements: "<<ker.nelements()
    //      <<"\toutput_elements: "<<out.nelements()
    //      << "\nin0:"<<in.dim(0)<<"\n in1:"<< in.dim(1)<<"\n in2:"<< in.dim(2)
    //      <<"\nkernel_n: "<<out.dim(0) *fanin<<"\n ker.dim1:"<<ker.dim(1)<<"\n ker.dim2:"
    //      <<ker.dim(2)
    //      <<"\nfanin:" <<fanin <<endl;
    // sync any previous kernel exec
    hipDeviceSynchronize();
    LOCAL_TIMING_REPORT("convgpu initial transfer time" << in.dim(0));
    LOCAL_TIMING2_START();
    if ((ker.dim(2) == 3) && (ker.dim(1) == 3))
      conv2mapgeneric <false, 3, 3> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 5) && (ker.dim(1) == 5))
      conv2mapgeneric <false, 5, 5> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 7) && (ker.dim(1) == 7))
      conv2mapgeneric <false, 7, 7> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 9) && (ker.dim(1) == 9))
      conv2mapgeneric <false, 9, 9> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 11) && (ker.dim(1) == 11))
      conv2mapgeneric <false, 11, 11> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 13) && (ker.dim(1) == 13))
      conv2mapgeneric <false, 13, 13> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 4) && (ker.dim(1) == 4))
      conv2mapgeneric <false, 4, 4> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 6) && (ker.dim(1) == 6))
      conv2mapgeneric <false, 6, 6> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 8) && (ker.dim(1) == 8))
      conv2mapgeneric <false, 8, 8> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                        in.dim(0), in.dim(1), in.dim(2),
                                                        out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                        stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 10) && (ker.dim(1) == 10))
      conv2mapgeneric <false, 10, 10> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y, table_data, fanin);
    else if ((ker.dim(2) == 12) && (ker.dim(1) == 12))
      conv2mapgeneric <false, 12, 12> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                          in.dim(0), in.dim(1), in.dim(2),
                                                          out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                          stride_x, stride_y, table_data, fanin);
    else
      conv2mapgeneric <false, 0 , 0> <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                                         in.dim(0), in.dim(1), in.dim(2),
                                                         out.dim(0)*fanin, ker.dim(1), ker.dim(2),
                                                         stride_x, stride_y, table_data, fanin);



  LOCAL_TIMING2_REPORT("convgpu kernel execution time");
  // sync & clean
  hipDeviceSynchronize();
  hipMemcpy(out.idx_ptr(), output_data, out.nelements() * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(input_data);
  hipFree(kernel_data);
  hipFree(output_data);
  hipFree(table_data);

  // check for errors
  err = hipGetLastError();
  if (err != hipSuccess)
    eblerror("cuda error: " << hipGetErrorString(err));
}



#endif

} // end namespace ebl
